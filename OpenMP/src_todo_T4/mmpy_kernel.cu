#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"

using namespace std;

#include <stdio.h>

// #define NAIVE

#ifdef NAIVE
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {

    int I =  blockIdx.y * blockDim.y + threadIdx.y;
    int J =  blockIdx.x * blockDim.x + threadIdx.x;

    if((I < N) && (J < N)){
        _FTYPE_ _c = 0;
        for (unsigned int k = 0; k < N; k++) {
            _FTYPE_ a = A[I * N + k];
            _FTYPE_ b = B[k * N + J];
            _c += a * b;
        }
        C[I * N + J] = _c;
    }
}
#else
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) { 
    int tx = threadIdx.x, ty = threadIdx.y;   // Built-in variable indicating the position of the current thread in the thread Block.
    int bx = blockIdx.x, by = blockIdx.y;     // Built-in variable to indicate the position of the current thread Block in the Grid.
    int bm_x = blockDim.x, bm_y = blockDim.y; // Represents the number of threads per Block.

    int row = by * bm_y * TILESCALE_M + ty;   // Calculate the position index of the current thread in the global matrix
    int col = bx * bm_x * TILESCALE_N + tx;   
    extern __shared__ _FTYPE_ sharedmem[];

    //__shared__ _FTYPE_ As[TILEDIM_M][TILEDIM_K];
    //__shared__ _FTYPE_ Bs[TILEDIM_K][TILEDIM_N];
    _FTYPE_ *As = (_FTYPE_ *)sharedmem;
    _FTYPE_ *Bs = (_FTYPE_ *)sharedmem + TILEDIM_M * TILEDIM_K;
    register _FTYPE_ C_saved[TILESCALE_M * TILESCALE_N] = {0.0f};

    int xIter = (TILEDIM_K + bm_x - 1) / bm_x; // This result indicates that in the x direction, each thread block needs to be divided into xIter small blocks for iteration.
    int yIter = (TILEDIM_K + bm_y - 1) / bm_y; 

    #pragma unroll
    for (int kk = 0; kk < N; kk += TILEDIM_K) {
        
        for (int xx = 0; xx < xIter; xx++)         
            for (int yy = 0; yy < TILESCALE_M; yy += 2) { 
                int a_row = ty + yy * bm_y;
                int a_col = tx + xx * bm_x; 
                int A_row = row + yy * bm_y;
                int A_col = kk + a_col;

                _FTYPE_ Aval = (A_col < N && A_row < N) ? A[A_row * N + A_col] : 0;
                As[a_row * TILEDIM_K + a_col] = Aval;

                a_row += bm_y;
                A_row += bm_y;

                Aval = (A_col < N && A_row < N) ? A[A_row * N + A_col] : 0;
                As[a_row * TILEDIM_K + a_col] = Aval;
            }
            
        __syncthreads();

        for (int xx = 0; xx < yIter; xx++)        
            for (int yy = 0; yy < TILESCALE_N; yy += 2) { 
                int b_row = ty + xx * bm_y;
                int b_col = tx + yy * bm_x; 
                int B_row = kk + b_row;
                int B_col = col + yy * bm_x;

                _FTYPE_ Bval = (B_col < N && B_row < N) ? B[B_row * N + B_col] : 0;
                Bs[b_row * TILEDIM_N + b_col] = Bval;

                b_col += bm_x;
                B_col += bm_x;

                Bval = (B_col < N && B_row < N) ? B[B_row * N + B_col] : 0;
                Bs[b_row * TILEDIM_N + b_col] = Bval;
            }

        __syncthreads();

        for (int k = 0; k < TILEDIM_K; k++) {
            #pragma unroll
            for (int j = 0; j < TILESCALE_N; j++) { // The value of Bs is preloaded to avoid repeated access in the inner loop
                _FTYPE_ bs_value = Bs[k * TILEDIM_N + tx + j * bm_x];
                #pragma unroll
                for (int i = 0; i < TILESCALE_M; i++) {
                    _FTYPE_ a_value = As[(ty + i * bm_y) * TILEDIM_K + k];
                    C_saved[i * TILESCALE_N + j] += a_value * bs_value;
                }
            }
        }
        
        __syncthreads();
    }

    // The result is written back to global memory
    #pragma unroll
    for (int j = 0; j < TILESCALE_N; j++) {
        #pragma unroll
        for (int i = 0; i < TILESCALE_M; i++) {
            int C_row = row + i * bm_y;
            int C_col = col + j * bm_x;
            if (C_row < N && C_col < N) {
                C[C_row * N + C_col] = C_saved[i * TILESCALE_N + j];
            }
        }
    }

}
#endif
